// gpu (device) based matrix/matrix gpu code
//-------------------------------------------------------------------------
// Included CUDA libraries
//-------------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <stdio.h>

// iceil macro
// returns an integer ceil value where integer numerator is first parameter
// and integer denominator is the second parameter. iceil is the rounded
// up value of numerator/denominator when there is a remainder
// equivalent to ((num%den!=0) ? num/den+1 : num/den)
#define iceil(num,den) (num+den-1)/den 

#define TILE_WIDTH 16 // block x and y dimensions

void check_error(hipError_t error_id){
   if (error_id != hipSuccess) {
      printf("Error is %d", error_id);
      exit(EXIT_FAILURE);
   }
}

// GPU device MatrixMulKernel kernel code 
__global__ void MatrixMulKernel(float *Pd, float *Md, float *Nd, int Mh,
   int Mw, int Nw) {
   // ==================================================================
   // Solution part 4
   // Determine the output index of each thread.
   // Compute the dot product of one row of Md and one column of Nd
   // for each thread.
   // Write the computed value to matrix P at the correct output index
   // ==================================================================

   // Calculate the global row and column indices of the Pd matrix
   int Row;
   int Col;
   //**** ENTER YOUR CODE HERE ****
   Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
   Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
   if(Row <Mh && Col< Nw){

   // Each thread computes one dot product element of the block sub-matrix
   // access correct row of Md and Column of Nd assuming row-major allocations
   // (Note: in second part of hw1 you will want to make sure that only
   //  the threads that are assigned valid regions of the computation are
   //  active
   //**** ENTER YOUR CODE HERE ****

   float Pvalue = 0;
   //**** ENTER YOUR CODE HERE ****
   for(int k=0; k < Mw ;++k){
      Pvalue += Md[Row*Mw+k] * Nd[k*Nw + Col];
   }

   // place final result in specified location of global Pd memory
   //**** ENTER YOUR CODE HERE ****
   Pd[Row * Nw + Col] = Pvalue;
   }

   // End of solution part 4 ===========================================
}

__global__ void MatrixMulKernelSingleBlock(float *Pd, float *Md, float *Nd, int Mh,
   int Mw, int Nw) {
   // ==================================================================
   // Solution part 4
   // Determine the output index of each thread.
   // Compute the dot product of one row of Md and one column of Nd
   // for each thread.
   // Write the computed value to matrix P at the correct output index
   // ==================================================================

   // Calculate the global row and column indices of the Pd matrix
   int Row;
   int Col;
   //**** ENTER YOUR CODE HERE ****
   Row = threadIdx.y;
   Col = threadIdx.x;
   if(Row > Mh || Col > Nw) return;

   // Each thread computes one dot product element of the block sub-matrix
   // access correct row of Md and Column of Nd assuming row-major allocations
   // (Note: in second part of hw1 you will want to make sure that only
   //  the threads that are assigned valid regions of the computation are
   //  active
   //**** ENTER YOUR CODE HERE ****

   float Pvalue = 0;
   //**** ENTER YOUR CODE HERE ****
   for(int k=0; k < Mw ;++k){
      Pvalue += Md[Row*Mw+k] * Nd[k*Nw + Col];
   }

   // place final result in specified location of global Pd memory
   //**** ENTER YOUR CODE HERE ****
   Pd[Row * Nw + Col] = Pvalue;

   // End of solution part 4 ===========================================
}

void compute_GPU(float *P, float *M, float *N, int Mh, 
   int Mw, int Nw) {
   float *Md, *Nd, *Pd;
   hipError_t error_id;

   // ===================================================================
   // Solution part 1: Copy Input Data from Host to Device
   //    Create Device Buffers for the two input matrices
   //    Copy memory from the host memory to the device buffer (device memory)
   //    Check for error generated while using each OpenCL API call
   // ===================================================================


   // Allocate device memory and Transfer host arrays M and N 
   //**** ENTER YOUR CODE HERE ****
   size_t size_M = Mh * Mw * sizeof(float);
   size_t size_N =  Mw*Nw * sizeof(float);
   error_id = hipMalloc((void**)&Md, size_M);
   check_error(error_id);
   error_id = hipMemcpy(Md, M, size_M, hipMemcpyHostToDevice);
   check_error(error_id);

   error_id = hipMalloc((void**)&Nd,size_N);
   check_error(error_id);
   error_id = hipMemcpy(Nd, N, size_N, hipMemcpyHostToDevice);
   check_error(error_id);

   // Allocate device memory of P array for results
   //**** ENTER YOUR CODE HERE ****
   size_t size_P = Mh*Nw*sizeof(float);
   error_id = hipMalloc((void**)&Pd, size_P );
   check_error(error_id);

   // End of solution Part 1 ============================================


   // ===================================================================
   // Solution part 2
   //    A. Initialize the block and grid dimensions of the kernel about
   //       to be launched.
   //       [You may assume that each matrix dimension is a multiple of the
   //        defined constant block_size.]
   //    B. Launch the kernel with appropriate kernel arguments
   //    Do not forget to check for success at each stage before proceeding.
   // ===================================================================

   // Setup the kernel execution configuration parameters/launch kernel

   // Stage A:  Setup the kernel execution configuration parameters
   //           (in second part of homework take into account the case where
   //            the dimmensions are not an even multiple of block size)
   //**** ENTER YOUR CODE HERE ****

   // Stage B: Launch the kernel!! -- using the appropriate function arguments
   //         (remember to check for kernel launch failure!)
   //**** ENTER YOUR CODE HERE ****
   if(Mh == 16 && Mw == 16 && Nw == 16){
      // Single Block 16*16 testing
      dim3 grid(1,1);
      dim3 block(TILE_WIDTH, TILE_WIDTH);
      MatrixMulKernelSingleBlock<<<grid,block>>>(Pd, Md, Nd,Mh, Mw, Nw);
   }
   else
   {
      int a1 = iceil(Mh, TILE_WIDTH);
      int a2 = iceil(Nw, TILE_WIDTH);
      dim3 grid(a2,a1);
      dim3 block(TILE_WIDTH, TILE_WIDTH);
      MatrixMulKernel<<<grid,block>>>(Pd, Md, Nd,Mh, Mw, Nw);
   }
   // End of solution Part 2 ============================================


   // ===================================================================
   // Solution part 3
   // Copy Results Device back to Host
   // ===================================================================

   // Transfer P from device to host
   //**** ENTER YOUR CODE HERE ****
   error_id = hipMemcpy(P,Pd,size_P,hipMemcpyDeviceToHost);
   check_error(error_id);


   // End of solution Part 3 ============================================


   // CLEAN UP -- Free device memory when finished
   //**** ENTER YOUR CODE HERE ****
   error_id = hipFree(Md);
   check_error(error_id);
   error_id = hipFree(Nd);
   check_error(error_id);
   error_id = hipFree(Pd);
   check_error(error_id);

}
